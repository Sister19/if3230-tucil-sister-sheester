// Cuda Parallel 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define MAX_N 512
#define BLOCK_SIZE 16

struct Matrix {
    int size;
    double mat[MAX_N][MAX_N];
};

struct FreqMatrix {
    int size;
    hipDoubleComplex mat[MAX_N][MAX_N];
};

__device__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
    double factor = exp(x.x);
    return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}

__global__ void dft_kernel(struct Matrix *mat, struct FreqMatrix *freq_domain)
{
  // Implement shared memory
    __shared__ double shared_mat[BLOCK_SIZE][BLOCK_SIZE];

    int k = blockIdx.x;
    int l = threadIdx.x;

    hipDoubleComplex element = make_hipDoubleComplex(0.0, 0.0);

    for (int i = 0; i < mat->size; i += BLOCK_SIZE) {
        for (int j = 0; j < mat->size; j += BLOCK_SIZE) {
            // Load a block of input matrix into shared memory
            shared_mat[l][k] = mat->mat[i + l][j + k];

            __syncthreads();

            for (int m = 0; m < BLOCK_SIZE; m++) {
                for (int n = 0; n < BLOCK_SIZE; n++) {
                    hipDoubleComplex arg      = make_hipDoubleComplex((i + m) * k / (double) mat->size + (j + n) * l / (double) mat->size, 0.0);
                    hipDoubleComplex exponent = cuCexp(make_hipDoubleComplex(0.0, -2.0 * M_PI * arg.x));
                    hipDoubleComplex value    = make_hipDoubleComplex(shared_mat[m][k], 0.0);
                    element = hipCadd(element, hipCmul(value, exponent));
                }
            }

            __syncthreads();
        }
    }

    element = hipCdiv(element, make_hipDoubleComplex(mat->size*mat->size, 0.0));
    freq_domain->mat[k][l] = element;
}

void readMatrix(struct Matrix *m)
{
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i][j]));
}

int main(void)
{
    struct Matrix source;
    struct FreqMatrix freq_domain;
    readMatrix(&source);
    freq_domain.size = source.size;

    // Allocate device memory
    struct Matrix *d_source;
    hipMalloc(&d_source, sizeof(struct Matrix));
    hipMemcpy(d_source, &source, sizeof(struct Matrix), hipMemcpyHostToDevice);

    struct FreqMatrix *d_freq_domain;
    hipMalloc(&d_freq_domain, sizeof(struct FreqMatrix));
    hipMemcpy(d_freq_domain, &freq_domain, sizeof(struct FreqMatrix), hipMemcpyHostToDevice);

    // Launch kernel
    dft_kernel<<<source.size, source.size>>>(d_source, d_freq_domain);

    // Copy results back to host
    hipMemcpy(&freq_domain, d_freq_domain, sizeof(struct FreqMatrix), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_source);
    hipFree(d_freq_domain);

    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    for (int k = 0; k < source.size; k++) {
        for (int l = 0; l < source.size; l++) {
            hipDoubleComplex el = freq_domain.mat[k][l];
            printf("(%lf, %lf) ", el.x, el.y);
            sum = hipCadd(sum, el);
        }
        printf("\n");
    }
    return 0;
};


// //alternatives

// #include <stdio.h>
// #include <stdlib.h>
// #include <math.h>
// #include <complex.h>

// #define MAX_N 512

// struct Matrix {
//     int    size;
//     double mat[MAX_N][MAX_N];
// };

// struct FreqMatrix {
//     int    size;
//     double complex mat[MAX_N][MAX_N];
// };

// __device__ double complex dft_element(struct Matrix *mat, int k, int l, int m, int n) {
//     double complex arg      = (k*m / (double) mat->size) + (l*n / (double) mat->size);
//     double complex exponent = cexp(-2.0I * M_PI * arg);
//     return mat->mat[m][n] * exponent;
// }

// __global__ void dft_kernel(struct Matrix *mat, struct FreqMatrix *freq_domain) {
//     int k = blockIdx.x * blockDim.x + threadIdx.x;
//     int l = blockIdx.y * blockDim.y + threadIdx.y;

//     if (k < freq_domain->size && l < freq_domain->size) {
//         double complex element = 0.0;
//         for (int m = 0; m < mat->size; m++) {
//             for (int n = 0; n < mat->size; n++) {
//                 element += dft_element(mat, k, l, m, n);
//             }
//         }
//         freq_domain->mat[k][l] = element / (double) (mat->size*mat->size);
//     }
// }

// void readMatrix(struct Matrix *m) {
//     scanf("%d", &(m->size));
//     for (int i = 0; i < m->size; i++)
//         for (int j = 0; j < m->size; j++)
//             scanf("%lf", &(m->mat[i][j]));
// }

// int main(void) {
//     struct Matrix     source;
//     struct FreqMatrix freq_domain;

//     readMatrix(&source);
//     freq_domain.size = source.size;

//     // Allocate memory on the device
//     struct Matrix     *d_source;
//     struct FreqMatrix *d_freq_domain;
//     cudaMalloc(&d_source, sizeof(struct Matrix));
//     cudaMalloc(&d_freq_domain, sizeof(struct FreqMatrix));

//     // Copy the data to the device
//     cudaMemcpy(d_source, &source, sizeof(struct Matrix), cudaMemcpyHostToDevice);

//     // Launch the kernel to compute the DFT
//     int block_size = 16;
//     dim3 dimBlock(block_size, block_size);
//     dim3 dimGrid(ceil(freq_domain.size / (float) block_size), ceil(freq_domain.size / (float) block_size));
//     dft_kernel<<<dimGrid, dimBlock>>>(d_source, d_freq_domain);

//     // Copy the results back to the host
//     cudaMemcpy(&freq_domain, d_freq_domain, sizeof(struct FreqMatrix), cudaMemcpyDeviceToHost);

//     double complex sum = 0.0;
//     for (int k = 0; k < source.size; k++) {
//         for (int l = 0; l < source.size; l++) {
//             double complex el = freq_domain.mat[k][l];
//             printf("(%lf, %lf) ", creal(el), cimag(el));
//             sum += el;
//         }
//         printf("\n");
//     }
//     sum /= source.size;
//     printf("Average : (%lf, %lf)", creal(sum), cimag(sum));

//     return 0;
// }
