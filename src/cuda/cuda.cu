
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_N 512

struct Matrix {
    int    size;
    double mat[MAX_N][MAX_N];
};

struct FreqMatrix {
    int    size;
    hipDoubleComplex mat[MAX_N][MAX_N];
};

hipDoubleComplex cuCexp(hipDoubleComplex x)
{
 double factor = exp(x.x);
 return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}

void readMatrix(struct Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i][j]));
}

hipDoubleComplex dft(struct Matrix *mat, int k, int l) {
    hipDoubleComplex element = make_hipDoubleComplex(0.0, 0.0);
    for (int m = 0; m < mat->size; m++) {
        for (int n = 0; n < mat->size; n++) {
            hipDoubleComplex arg      = make_hipDoubleComplex(k*m / (double) mat->size + l*n / (double) mat->size, 0.0);
            hipDoubleComplex exponent = cuCexp(make_hipDoubleComplex(0.0, -2.0 * M_PI * arg.x));
            hipDoubleComplex value = make_hipDoubleComplex(mat->mat[m][n], 0.0);
            element = hipCadd(element, hipCmul(value, exponent));
        }
    }
    element = hipCdiv(element, make_hipDoubleComplex(mat->size*mat->size, 0.0));
    return element;
}

int main(void) {
    struct Matrix     source;
    struct FreqMatrix freq_domain;
    readMatrix(&source);
    freq_domain.size = source.size;
    
    for (int k = 0; k < source.size; k++)
        for (int l = 0; l < source.size; l++)
            freq_domain.mat[k][l] = dft(&source, k, l);

    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    for (int k = 0; k < source.size; k++) {
        for (int l = 0; l < source.size; l++) {
            hipDoubleComplex el = freq_domain.mat[k][l];
            printf("(%lf, %lf) ", el.x, el.y);
            sum = hipCadd(sum, el);
        }
        printf("\n");
    }
    sum = hipCdiv(sum, make_hipDoubleComplex(freq_domain.size, 0.0));
    printf("Average : (%lf, %lf)", sum.x, sum.y);

    return 0;
}